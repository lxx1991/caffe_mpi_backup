#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaskForward(const int nthreads, const Dtype threshold, int spatial_dim, const int channels,
    const Dtype* prob, Dtype* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    mask[index] = 1;
    for (int i=0; i<channels; i++)
      if (prob[i * spatial_dim + index] > threshold + 1e-9) mask[index] = 0;
  }
}


template <typename Dtype>
__global__ void MaskForward(const int nthreads, const Dtype threshold_easy, const Dtype threshold_hard, const int spatial_dim, const int channels,
    const Dtype* prob, Dtype* mask, const int ignore_label, const Dtype* label, Dtype* new_label) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    mask[index] = 1;
    const int label_value = static_cast<int>(label[index]);
    new_label[index] = label_value;
    if (label_value < channels && prob[label_value * spatial_dim + index] > threshold_easy + 1e-9)
    {
      mask[index] = 0;
      new_label[index] = ignore_label;
    }
    else
    {
      for (int i = 0; i<channels; i++)
        if (prob[i * spatial_dim + index] > threshold_hard + 1e-9)
        {
          mask[index] = 0;
          new_label[index] = ignore_label;
        }
    }
  }
}

template <typename Dtype>
void MaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* prob = bottom[0]->gpu_data();
  Dtype* mask = top[0]->mutable_gpu_data();
  const int count = top[0]->count();
  int spatial_dim = count;

  if (top.size() == 1)
  {
     MaskForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, threshold_easy_, spatial_dim, bottom[0]->channels(), prob, mask);
  }
  else
  {
      const Dtype* label_data = bottom[1]->gpu_data();
      Dtype* new_label = top[1]->mutable_gpu_data();

      MaskForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, threshold_easy_, threshold_hard_, spatial_dim, bottom[0]->channels(), prob, mask, ignore_label_, label_data, new_label);
  }

  
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void MaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
  
}


INSTANTIATE_LAYER_GPU_FUNCS(MaskLayer);


}  // namespace caffe
