#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaskForward(const int nthreads, const Dtype threshold, const bool has_negative_label, const int negative_label, int spatial_dim, const int channels,
    const Dtype* prob, Dtype* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    mask[index] = 1;
    if (has_negative_label)
    {
      if (prob[negative_label * spatial_dim + index] > threshold) mask[index] = 0;
    }
    else
    {
      for (int i=0; i<channels; i++)
        if (prob[i * spatial_dim + index] > threshold) mask[index] = 0;
    }
  }
}


template <typename Dtype>
__global__ void MaskForward(const int nthreads, const Dtype threshold, const bool has_negative_label, const int negative_label, const int spatial_dim, const int channels,
    const Dtype* prob, Dtype* mask, const int ignore_label, const Dtype* label, Dtype* new_label) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    mask[index] = 1;
    const int label_value = static_cast<int>(label[index]);
    new_label[index] = label_value;

    if (has_negative_label)
    {
      if (prob[negative_label * spatial_dim + index] > threshold && label_value == negative_label)
      {
        mask[index] = 0;
        new_label[index] = ignore_label;
      }
    }
    else
    {
      if (label_value < channels && prob[label_value * spatial_dim + index] > threshold)
      {
        mask[index] = 0;
        new_label[index] = ignore_label;
      }
    }
  }
}

template <typename Dtype>
void MaskLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* prob = bottom[0]->gpu_data();
  Dtype* mask = top[0]->mutable_gpu_data();
  const int count = top[0]->count();
  int spatial_dim = count;

  if (top.size() == 1)
  {
     MaskForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, threshold_, has_negative_label_, negative_label_, spatial_dim, bottom[0]->channels(), prob, mask);
  }
  else
  {
      const Dtype* label_data = bottom[1]->gpu_data();
      Dtype* new_label = top[1]->mutable_gpu_data();

      MaskForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, threshold_, has_negative_label_, negative_label_, spatial_dim, bottom[0]->channels(), prob, mask, ignore_label_, label_data, new_label);
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void MaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
  
}


INSTANTIATE_LAYER_GPU_FUNCS(MaskLayer);


}  // namespace caffe
