#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SpatialFilter(const int nthreads, const int channels, const int spatial_dim, const Dtype* data, const Dtype* mask, Dtype* data_out) {
  CUDA_KERNEL_LOOP(index, nthreads) {
      for (int i=0; i<channels; i++)
      {
        if (mask[index] < 0.5)
          data_out[i * spatial_dim + index] = 0;
        else
          data_out[i * spatial_dim + index] = data[i * spatial_dim + index];
      }
  }
}


template <typename Dtype>
void SpatialFilterLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  
  const Dtype* mask = bottom[0]->gpu_data();
  const Dtype* data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int spatial_dim = count;

  SpatialFilter<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom[1]->channels(), spatial_dim, data, mask, top_data);

  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void SpatialFilterLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
  const Dtype* mask = bottom[0]->gpu_data();
  const Dtype* data = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[1]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  const int spatial_dim = count;

  SpatialFilter<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom[1]->channels(), spatial_dim, data, mask, bottom_diff);
  
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(SpatialFilterLayer);


}  // namespace caffe
